
#include <hip/hip_runtime.h>
// The chirp matrix is here 
typedef float2 Complex; 
#define TWOPI 6.2831853071796 
#define DFFAC 2.41e-10 /*DM (pc cm-3) = DFFAC*D (MHz) */
// Very Very Bad Programming Practise Here
#define fsky 1300.3333E6f // Mhz
#define bw 120E6f // Mhz
#define sideband 1

__global__ void ddtchirp(Complex * chirp, float delta, long N) {
		delta = DFFAC * delta;
		int n, l, k;
		n = blockIdx.x * blockDim.x + threadIdx.x;
		l = blockIdx.y * blockDim.y + threadIdx.y;
		k = threadIdx.z;
		//
		float kappa, taper, f;
		f = (n - N/2) * bw / (N-1);
		f += fsky;
		if(f > 0.5 * bw)
				taper = 1.0 / sqrt(1.0 + pow( (f/(0.94 * bw)), 80));
		else
				taper = 1.0 / sqrt(1.0 + pow( ((bw -f)/(0.94 * bw)), 80));
		if(n <= N/2)
				kappa = -1 * delta * f * f /(f + f*fsky);
		else
				kappa = delta * f * f /(f + f*fsky);
		//
		/*
		 *TWOPI * ( k * (n-l)/N) 
		 *TWOPI * ( kappa)
		 */
		chirp[n * N + l].x += taper * cos ( TWOPI * ( ( k * (n-l)/N) + kappa ));
		chirp[n * N + l].y -= taper * sin ( TWOPI * ( ( k * (n-l)/N) + kappa ));
}

__global__ void fftchirp(Complex * chirp, float delta, long N){
		delta = DFFAC * delta;
		int n;
		n = blockIdx.x * blockDim.x + threadIdx.x;
		//
		float kappa, taper, f;
		f = (n - N/2) * bw / (N-1);
		f += fsky;
		if(f > 0.5 * bw)
				taper = 1.0 / sqrt(1.0 + pow( (f/(0.94 * bw)), 80));
		else
				taper = 1.0 / sqrt(1.0 + pow( ((bw -f)/(0.94 * bw)), 80));
		if(n <= N/2)
				kappa = -1 * delta * f * f /(f + f*fsky);
		else
				kappa = delta * f * f /(f + f*fsky);
		//
		/*
		 *TWOPI * ( k * (n-l)/N) 
		 *TWOPI * ( kappa)
		 *****************
		 * Negative to take inverse filter
		 */
		chirp[n].x += taper * cos ( TWOPI * ( kappa ));
		chirp[n].y -= taper * sin ( TWOPI * ( kappa ));
}

__device__ Complex ComplexMult(Complex one, Complex two) {
		Complex ret; 
		ret.x = one.x * two.x - one.y * two.y;
		ret.y = one.x * two.y + one.y * two.x;
		return ret;
}

__global__ void vecpro(Complex * i1, Complex * i2, Complex * out){
		int n;
		n = blockIdx.x * blockDim.x + threadIdx.x;
		out[n] = ComplexMult(i1[n],i2[n]); 
}
