// This is Gamma.
// This is the original code

// Constants
#define TWOPI 6.2831853071796 
#define DFFAC 2.41e-10 /*DM (pc cm-3) = DFFAC*D (MHz) */
// Very Very Bad Programming Practise Here
#define fsky 1300.3333E6f // Mhz
#define bw 120E6f // Mhz
#define sideband 1

// Standard includes
#include "stdio.h"
#include "string.h"
#include "stdlib.h"
#include "math.h"

// CUDA includes
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "hipblas.h"
//#include <helper_cuda.h>


// typedef 
// because I am lazy to write float2 everywhere
typedef float2 Complex; 

// forward declarations 
__global__ void ddtchirp(Complex * chirp, float delta, long N);
__global__ void fftchirp(Complex * chirp, float delta, long N);
__device__ Complex ComplexMult(Complex one, Complex two);
__global__ void vecpro(Complex * i1, Complex * i2, Complex * out);
#define checkCudaErrors(cce) {\
		hipError_t cer = cce;\
		if(cer != hipSuccess) {\
				printf("[!!] CUDA Failure at %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(cer));\
				exit(1);\
		}\
}

// starting Main
int main(int argc, char * argv[]){
		if(argc < 2) {
				printf("Direct De-dispersion Transform\n");
				printf("Usage : <program> <DM> <exponent of two> <filename>\n");
				printf("Written by Suryarao Bethapudi[ep14btech11008@iith.ac.in]\n");
				return 0;
		}
		// check for CUDA
		int devID = 0; // Device ID
		hipDeviceProp_t deviceProp;
		checkCudaErrors(hipSetDevice(devID));
		checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
		/*printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);*/
		// basic part
		//
		long N = (long)strtol(argv[2],NULL,10);
		double dm;
		FILE * fp;
		fp = fopen((const char*)argv[3],"r");
		N = pow(2,N);
		dm = (double)strtod(argv[1],NULL);

		// complex pointers for host and device memory
		Complex * h_in, * d_in;
		Complex * ddt_out, * fft_out;
		// NOTE: In some future implementation, 
		// copy de-dispersed back to host and do error analysis
		Complex * ddtchirp_mat;
		long memsize = N * sizeof(Complex);	
		// Allocating in Host
		h_in = (Complex*)malloc(memsize);
		/*h_out = (Complex*)malloc(memsize);*/
		// FILE IO 
		long i;
		float dx;
		Complex t;
		for(i = 0; i < N;i++){
				fscanf(fp,"%f\n",&dx);
				t.x = dx;
				t.y = 0.0f;
				h_in[i] = t;
		}
		///////////////////////////////////////////////////////////////////
		// Allocating in Device
		// NOTE: For now, I am creating three N-arrays. 
		checkCudaErrors(hipMalloc((void**)&d_in,memsize));
		checkCudaErrors(hipMalloc((void**)&ddt_out,memsize));
		checkCudaErrors(hipMalloc((void**)&ddtchirp_mat,N*memsize)); 
		// Need N^2 elements
		checkCudaErrors(hipMalloc((void**)&fft_out,memsize));
		// These are compressed commands

		// Copying from Host to Device 
		checkCudaErrors(hipMemcpy(d_in, h_in, memsize, hipMemcpyHostToDevice));

		// cuda variables, types
		// measure time
		hipEvent_t estart, estop; 
		checkCudaErrors( hipEventCreate(&estart) );
		checkCudaErrors( hipEventCreate(&estop) );
		// elasped time
		float t_fftchirp, t_fft, t_ddtchrip, t_ddt; 
		// plans and handles	
		hipblasStatus_t cstat;
		hipfftHandle cplan;
		hipblasHandle_t candle;
		hipfftResult cufftres;
		cstat = hipblasCreate(&candle); // creating handle
		if(cstat != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr,"CUBLAS Initialization failed...\n");
				return 1;
		}
		///////////////////////////////////////////////////////////////////
		//The FFT heart
		cufftres = hipfftPlan1d(&cplan, N, HIPFFT_C2C, 1);
		if(cufftres != HIPFFT_SUCCESS) {
				fprintf(stderr,"CUFFT Error: Plan creation failed!.\n");
				return 1;
		}
		checkCudaErrors( hipEventRecord(estart,0) );
		// This is the actual kernel call
		dim3 grid1(4,4,4);
		dim3 block1(32,32,1);
		fftchirp<<<grid1,block1>>>(fft_out, dm, N);
		checkCudaErrors( hipEventRecord(estop,0) );
		checkCudaErrors( hipEventSynchronize(estop) );
		checkCudaErrors( hipEventElapsedTime(&t_fftchirp, estart, estop) );
		// FFT CHIRP timed
		////////////////////////////////////////////////////////////////////
		checkCudaErrors( hipEventRecord(estart,0) );
		cufftres = hipfftExecC2C(cplan, (hipfftComplex*)d_in, (hipfftComplex*)fft_out, HIPFFT_FORWARD); 
		if(cufftres != HIPFFT_SUCCESS) {
				fprintf(stderr,"CUFFT Error: Transform failed!.\n");
				return 1;
		}
		dim3 grid2(4,4,4);
		dim3 block2(32,32,1);
		vecpro<<<grid2,block2>>>(fft_out,d_in,fft_out);
		cufftres = hipfftExecC2C(cplan,(hipfftComplex*)fft_out, (hipfftComplex*)fft_out, HIPFFT_BACKWARD); 
		if(cufftres != HIPFFT_SUCCESS) {
				fprintf(stderr,"CUFFT Error: Transform failed!.\n");
				return 1;
		}
		checkCudaErrors( hipEventRecord(estop,0) );
		checkCudaErrors( hipEventSynchronize(estop) );
		checkCudaErrors( hipEventElapsedTime(&t_fft, estart, estop) );
		// FFT timed 
		//////////////////////////////////////////////////////////////////

		//////////////////////////////////////////////////////////////////
		// The DDT heart.
		checkCudaErrors( hipEventRecord(estart,0) );
		dim3 grid3(4,4,4);
		dim3 block3(32,32,1);
		ddtchirp<<<grid3,block3>>>(ddtchirp_mat,dm,N);
		checkCudaErrors( hipEventRecord(estop,0) );
		checkCudaErrors( hipEventSynchronize(estop) );
		checkCudaErrors( hipEventElapsedTime(&t_ddtchrip, estart, estop) );
		// DDT chirp timed 
		//////////////////////////////////////////////////////////////////
		t.x = 1.0f;
		t.y = 0.0f;
		checkCudaErrors( hipEventRecord(estart,0) );
		Complex u;
		u.x = 0.0f;
		u.y = 0.0f;
		cstat = hipblasCgemv(candle, HIPBLAS_OP_N, N, N, &t, (hipComplex*)ddtchirp_mat, N, (hipComplex*)d_in, 1, &u, (hipComplex*)ddt_out, 1);
		if(cstat != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr,"CUBLAS Error: GEMV failed!.\n");
				return 1;
		}
		checkCudaErrors( hipEventRecord(estop,0) );
		checkCudaErrors( hipEventSynchronize(estop) );
		checkCudaErrors( hipEventElapsedTime(&t_ddt, estart, estop) );
		// DDT timed 
		//////////////////////////////////////////////////////////////////

		// Blocking 
		if(hipDeviceSynchronize() != hipSuccess){
				fprintf(stderr,"CUDA Error: Failed to synchronize..\n");
				return 1;
		}
		// Compute MSE
		// Result goes in fft_out
		cstat = hipblasCaxpy(candle, N, &t, ddt_out, 1, fft_out, 1);
		if(cstat != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr,"CUBLAS Error: AXPY failed!.\n");
				return 1;
		}
		float mse;
		cstat = hipblasScnrm2(candle, N, fft_out, 1, &mse);
		if(cstat != HIPBLAS_STATUS_SUCCESS) {
				fprintf(stderr,"CUBLAS Error: NRM2 failed!.\n");
				return 1;
		}
		/////////////////////////////////////////////////////////////////
		printf("%ld, %f, %f. %f, %f, %f\n",N, t_fftchirp, t_ddtchrip, t_fft, t_ddt, mse);
		// Exit
		hipEventDestroy(estart); // destroying events
		hipEventDestroy(estop);
		hipfftDestroy(cplan); // destorying plan 
		hipblasDestroy(candle); // destorying handle
		// free 
		free(h_in);
		checkCudaErrors( hipFree(fft_out) );
		checkCudaErrors( hipFree(ddt_out) );
		checkCudaErrors( hipFree(d_in) );
		checkCudaErrors( hipFree(ddtchirp_mat) );
		return 0;
}

__global__ void ddtchirp(Complex * chirp, float delta, long N) {
		delta = DFFAC * delta;
		int n, l, k;
		n = blockIdx.x * blockDim.x + threadIdx.x;
		l = blockIdx.y * blockDim.y + threadIdx.y;
		k = threadIdx.z;
		//
		float kappa, taper, f;
		f = (n - N/2) * bw / (N-1);
		f += fsky;
		if(f > 0.5 * bw)
				taper = 1.0 / sqrt(1.0 + pow( (f/(0.94 * bw)), 80));
		else
				taper = 1.0 / sqrt(1.0 + pow( ((bw -f)/(0.94 * bw)), 80));
		if(n <= N/2)
				kappa = -1 * delta * f * f /(f + f*fsky);
		else
				kappa = delta * f * f /(f + f*fsky);
		//
		/*
		 *TWOPI * ( k * (n-l)/N) 
		 *TWOPI * ( kappa)
		 */
		chirp[n * N + l].x += taper * cos ( TWOPI * ( ( k * (n-l)/N) + kappa ));
		chirp[n * N + l].y -= taper * sin ( TWOPI * ( ( k * (n-l)/N) + kappa ));
}

__global__ void fftchirp(Complex * chirp, float delta, long N){
		delta = DFFAC * delta;
		int n;
		n = blockIdx.x * blockDim.x + threadIdx.x;
		//
		float kappa, taper, f;
		f = (n - N/2) * bw / (N-1);
		f += fsky;
		if(f > 0.5 * bw)
				taper = 1.0 / sqrt(1.0 + pow( (f/(0.94 * bw)), 80));
		else
				taper = 1.0 / sqrt(1.0 + pow( ((bw -f)/(0.94 * bw)), 80));
		if(n <= N/2)
				kappa = -1 * delta * f * f /(f + f*fsky);
		else
				kappa = delta * f * f /(f + f*fsky);
		//
		/*
		 *TWOPI * ( k * (n-l)/N) 
		 *TWOPI * ( kappa)
		 *****************
		 * Negative to take inverse filter
		 */
		chirp[n].x += taper * cos ( TWOPI * ( kappa ));
		chirp[n].y -= taper * sin ( TWOPI * ( kappa ));
}

__device__ Complex ComplexMult(Complex one, Complex two) {
		Complex ret; 
		ret.x = one.x * two.x - one.y * two.y;
		ret.y = one.x * two.y + one.y * two.x;
		return ret;
}

__global__ void vecpro(Complex * i1, Complex * i2, Complex * out){
		int n;
		n = blockIdx.x * blockDim.x + threadIdx.x;
		out[n] = ComplexMult(i1[n],i2[n]); 
}
