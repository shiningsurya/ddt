#include "hip/hip_runtime.h"
// The chirp matrix is here 
__global__ void ddtchirp(Complex * chirp, float delta, long N) {
		int n, l, k;
		n = blockIdx.x * TILE_WIDTH + threadIdx.x;
		l = blockIdx.y * TILE_WIDTH + threadIdx.y;
		k = threadIdx.z;
		//
		float kappa, taper, f;
		f = (n - N/2) * bw / (N-1);
		f += fsky;
		if(f > 0.5 * bw)
				taper = 1.0 / sqrt(1.0 + pow( (f/(0.94 * bw)), 80);
		else
				taper = 1.0 / sqrt(1.0 + pow( ((bw -f)/(0.94 * bw)), 80);
		if(n <= N/2)
				kappa = -1 * delta * f * f /(f + f*fsky);
		else
				kappa = delta * f * f /(f + f*fsky);
		//
		/*
		 *TWOPI * ( k * (n-l)/N) 
		 *TWOPI * ( kappa)
		 */
		chirp[n * N + l].x += taper * cos ( TWOPI * ( ( k * (n-l)/N) + kappa ));
		chirp[n * N + l].y -= taper * sin ( TWOPI * ( ( k * (n-l)/N) + kappa ));
}

__global__ void fftchirp(Complex * chirp, float delta, long N){
		int n;
		n = blockIdx.x + TILE_WIDTH + threadIdx.x;
		//
		float kappa, taper, f;
		f = (n - N/2) * bw / (N-1);
		f += fsky;
		if(f > 0.5 * bw)
				taper = 1.0 / sqrt(1.0 + pow( (f/(0.94 * bw)), 80);
		else
				taper = 1.0 / sqrt(1.0 + pow( ((bw -f)/(0.94 * bw)), 80);
		if(n <= N/2)
				kappa = -1 * delta * f * f /(f + f*fsky);
		else
				kappa = delta * f * f /(f + f*fsky);
		//
		/*
		 *TWOPI * ( k * (n-l)/N) 
		 *TWOPI * ( kappa)
		 *****************
		 * Negative to take inverse filter
		 */
		chirp[n].x += taper * cos ( TWOPI * ( kappa ));
		chirp[n].y -= taper * sin ( TWOPI * ( kappa ));
}

__device__ Complex ComplexMult(Complex one, Complex two) {
		Complex ret; 
		ret.x = one.x * two.x - one.y * two.y;
		ret.y = one.x * two.y + one.y * two.x;
		return ret;
}

__global__ void vecpro(Complex * i1, Complex * i2, Complex * out){
		int n;
		n = blockIdx.x * TILE_WIDTH + threadIdx.x;
		out[n] = ComplexMult(i1[n],i2[n]); 
}
