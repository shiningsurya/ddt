#include "hip/hip_runtime.h"
// The chirp matrix is here 

__global__
void ddtchirp(double * chirp1, double * chirp2, double delta, long long int N) {
		int n, l, k;
		n = blockIdx.x * TILE_WIDTH + threadIdx.x;
		l = blockIdx.y * TILE_WIDTH + threadIdx.y;
		k = threadIdx.z;
		//
		double kappa, taper, f;
		f = (n - N/2) * bw / (N-1);
		f += fsky;
		if(f > 0.5 * bw)
				taper = 1.0 / sqrt(1.0 + pow( (f/(0.94 * bw)), 80);
		else
				taper = 1.0 / sqrt(1.0 + pow( ((bw -f)/(0.94 * bw)), 80);
		if(n <= N/2)
				kappa = -1 * delta * f * f /(f + f*fsky);
		else
				kappa = delta * f * f /(f + f*fsky);
		//
		/*
		 *TWOPI * ( k * (n-l)/N) 
		 *TWOPI * ( kappa)
		 */
		chirp1[n * N + l] += taper * cos ( TWOPI * ( ( k * (n-l)/N) + kappa ));
		chirp2[n * N + l] -= taper * sin ( TWOPI * ( ( k * (n-l)/N) + kappa ));
}

__global__
void fftchirp(double * chirp1, double * chirp2, double delta, long long int N){
		int n;
		n = blockIdx.x + TILE_WIDTH + threadIdx.x;
		//
		double kappa, taper, f;
		f = (n - N/2) * bw / (N-1);
		f += fsky;
		if(f > 0.5 * bw)
				taper = 1.0 / sqrt(1.0 + pow( (f/(0.94 * bw)), 80);
		else
				taper = 1.0 / sqrt(1.0 + pow( ((bw -f)/(0.94 * bw)), 80);
		if(n <= N/2)
				kappa = -1 * delta * f * f /(f + f*fsky);
		else
				kappa = delta * f * f /(f + f*fsky);
		//
		/*
		 *TWOPI * ( k * (n-l)/N) 
		 *TWOPI * ( kappa)
		 *****************
		 * Negative to take inverse filter
		 */
		chirp1[n] += taper * cos ( TWOPI * ( kappa ));
		chirp2[n] -= taper * sin ( TWOPI * ( kappa ));
}

__global__
void vecpro(double * i1, double * i2, double * out){
		int n;
		n = blockIdx.x * TILE_WIDTH + threadIdx.x;
		out[n] = i1[n] * i2[n];
}
